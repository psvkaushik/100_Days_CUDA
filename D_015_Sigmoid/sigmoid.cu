#include <iostream>
#include <hip/hip_runtime.h>

#define N 10   // Size of the array

__global__ void sigmoid(float *input, float *output, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        output[idx] = 1.0f / (1.0f + expf(-input[idx]));  // Sigmoid function
    }
}

int main() {
    float h_input[N] = {-2.0, -1.0, 0.0, 1.0, 2.0, -3.0, 5.0, -4.0, 3.0, -0.5};
    float h_output[N] = {0};  // Initialize output array

    float *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with dynamic thread and block calculations
    int threadsPerBlock = 256;  // Standard thread block size
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;  // Round up to cover all elements
    sigmoid<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Sigmoid output:\n";
    for (int i = 0; i < N; i++) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}

#include <iostream>
#include <hip/hip_runtime.h>

#define N 10   // Size of the array

__global__ void relu(float *input, float *output, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < n) {
        output[idx] = fmaxf(0.0f, input[idx]);  // ReLU function
    }
}



int main() {
    float h_input[N] = {-2.0, -1.0, 0.0, 1.0, 2.0, -3.0, 5.0, -4.0, 3.0, -0.5};
    float h_output[N] = {0};  // Initialize output array

    float *d_input, *d_output;
    hipMalloc(&d_input, N * sizeof(float));
    hipMalloc(&d_output, N * sizeof(float));

    hipMemcpy(d_input, h_input, N * sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel with dynamic thread and block calculations
    int threadsPerBlock = 256;  // Standard thread block size
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;  // Round up to cover all elements
    relu<<<blocksPerGrid, threadsPerBlock>>>(d_input, d_output, N);
    hipDeviceSynchronize();

    hipMemcpy(h_output, d_output, N * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "ReLU output:\n";
    for (int i = 0; i < N; i++) {
        std::cout << h_output[i] << " ";
    }
    std::cout << std::endl;

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}

#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <hip/hip_runtime.h>

// CUDA Kernel for Online Softmax
__global__ void online_softmax_kernel(const float* input, float* output, int rows, int cols) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows) {
        float max_val = -FLT_MAX;
        float sum_exp = 0.0f;
        for (int i = 0; i < cols; i++) {
            if (max_val == -FLT_MAX){
                max_val = input[row*cols + i];
            }
            if (input[row * cols + i] > max_val && max_val != -FLT_MAX) {
                float correction_factor = max_val - input[row*cols + i];
                sum_exp *= expf(correction_factor);
                max_val = input[row*cols + i];
            }


            sum_exp += expf(input[row * cols + i] - max_val);
        }

        for (int i = 0; i < cols; i++) {
            output[row * cols + i] = expf(input[row * cols + i] - max_val) / sum_exp;
        }
    }
}

// Host function to launch the kernel
void online_softmax_launcher(const torch::Tensor& input, torch::Tensor& output) {
    const auto rows = input.size(0);
    const auto cols = input.size(1);

    const dim3 threads_per_block(256);
    const dim3 num_blocks((rows + threads_per_block.x - 1) / threads_per_block.x);

    online_softmax_kernel<<<num_blocks, threads_per_block>>>(
        input.data_ptr<float>(),
        output.data_ptr<float>(),
        rows,
        cols
    );
}

// PyTorch binding
PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("online_softmax", &online_softmax_launcher, "Online Softmax CUDA");
}

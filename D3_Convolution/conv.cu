#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

#define KERNEL_SIZE 3
#define Filter_Radius 1

__constant__ float kernel_gpu[KERNEL_SIZE][KERNEL_SIZE];

__global__ void conv_constant_kernel(float* input, float* output, int height, int width) {

    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    float temp = 0.0f;
    if (row < height && col<width) {
        
        for (int i=0; i<KERNEL_SIZE;i++){
            for (int j=0;j<KERNEL_SIZE;j++){
                int inRow = row - Filter_Radius +i;
                int inCol = col - Filter_Radius + j;
                if (inRow >=0 && inRow <height && inCol >=0 && inCol < width) {
                    temp += input[inRow*width + inCol] * kernel_gpu[i][j];
                }
            }
        }
    output[row*width + col] = temp;


    }
}

void conv_constant(float *input, int height, int width, float *kernel, float *output){
    float *input_gpu, *output_gpu;
    float size = height * width * sizeof(float);
    // Allocate the memory in GPU
    hipMalloc((void**) &input_gpu, size);
    hipMalloc((void **)&output_gpu, size);

    //copy contents from CPU to GPU

    hipMemcpy(input_gpu, input, size, hipMemcpyHostToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(kernel_gpu), kernel, KERNEL_SIZE*KERNEL_SIZE*sizeof(float));

    //Launch the kernel

    dim3 numThreads (32, 32);
    dim3 numBlocks ((width + numThreads.x - 1)/numThreads.x, (height + numThreads.y-1)/numThreads.y);

    conv_constant_kernel<<<numBlocks, numThreads>>>(input_gpu, output_gpu, height, width);

    // Copy contents of the output back to GPU
    
    hipMemcpy(output, output_gpu, size, hipMemcpyDeviceToHost);
    
    // free the space
    hipFree(output_gpu);
    hipFree(input_gpu);


}
// Function to perform 2D convolution on CPU
void convolution2DCPU(float* input, float* output, int width, int height, float* kernel) {
    for (int row = 0; row < height; row++) {
        for (int col = 0; col < width; col++) {
            float value = 0.0f;
            for (int i = 0; i < KERNEL_SIZE; i++) {
                for (int j = 0; j < KERNEL_SIZE; j++) {
                    int inputRow = row - KERNEL_SIZE / 2 + i;
                    int inputCol = col - KERNEL_SIZE / 2 + j;

                    if (inputRow >= 0 && inputRow < height && inputCol >= 0 && inputCol < width) {
                        value += input[inputRow * width + inputCol] * kernel[i * KERNEL_SIZE + j];
                    }
                }
            }
            output[row * width + col] = value;
        }
    }
}

// Function to print a 1D array as a 2D matrix
void printArray(float* arr, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << arr[i * cols + j] << " ";
        }
        std::cout << std::endl;
    }
}

// Function to compare two matrices
bool compareMatrices(float *c, float *c_gpu, int m, int n, float tolerance = 1e-4) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            if (fabs(c[i * n + j] - c_gpu[i * n + j]) > tolerance) {
                cout << "Mismatch at (" << i << ", " << j << "): CPU = " << c[i * n + j] << ", GPU = " << c_gpu[i * n + j] << " " << c[i * n + j] - c_gpu[i * n + j] << endl;
                return false;
            }
        }
    }
    return true;
}

int main() {
    int height = 320000;
    int width = 3200;
    int kernelHeight = 2 * Filter_Radius + 1;
    int kernelSize = kernelHeight * kernelHeight;

    float *input = (float*)malloc(height * width * sizeof(float));
    float *kernel = (float*)malloc(kernelSize * sizeof(float));
    float *output = (float*)malloc(height * width * sizeof(float));

    // Initialize input and kernel with random values
    for (int i = 0; i < height; i++) {
        for (int j = 0; j < width; j++) {
            input[i * width + j] = static_cast<float>(rand()) / RAND_MAX;
        }
    }

    for (int i = 0; i < kernelHeight; i++) {
        for (int j = 0; j < kernelHeight; j++) {
            kernel[i * kernelHeight + j] = static_cast<float>(rand()) / RAND_MAX;
        }
    }

    // Perform convolution on CPU
    clock_t start, stop;
    start = clock();
    convolution2DCPU(input, output, width, height, kernel);
    stop = clock();
    double cpu_time_used = (double)(stop - start) / CLOCKS_PER_SEC;
    cout << "The amount of time taken by the CPU is " << cpu_time_used * 1000 << " ms" << endl;


    // Perform convolution on GPU
    float *output_gpu = (float*)malloc(height * width * sizeof(float));
    hipEvent_t start_gpu, stop_gpu;
    hipEventCreate(&start_gpu);
    hipEventCreate(&stop_gpu);

    hipEventRecord(start_gpu);
    conv_constant(input, height, width, kernel, output_gpu);
    hipEventRecord(stop_gpu);
    hipEventSynchronize(stop_gpu);

    float gpu_time_used;
    hipEventElapsedTime(&gpu_time_used, start_gpu, stop_gpu);
    cout << "GPU Time: " << gpu_time_used << " ms" << endl;

    // Compare CPU and GPU outputs
    if (compareMatrices(output, output_gpu, height, width)) {
        std::cout << "Matrices match!" << std::endl;
    } else {
        std::cout << "Matrices do not match!" << std::endl;
    }

    // Free memory
    free(input);
    free(kernel);
    free(output);
    free(output_gpu);

    return 0;
}